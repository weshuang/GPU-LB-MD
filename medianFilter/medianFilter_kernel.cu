#include "hip/hip_runtime.h"
#include "medianFilter_kernel.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <cuPrintf.cu>
#include <hip/hip_runtime.h>

/* You can modify this file as you want. */
__global__ void mfKernel(const RGB* src, RGB* dst, int r, int w, int h);

//#define CCE checkCudaErrors




extern "C" void
medianFilter(
             const uchar *h_src,
             uchar *h_dst,
             int w,
             int h,
             int r)
{
	int block_size=16;
	dim3 blockDim(/*BLOCK_SIZE, BLOCK_SIZE*/16,16);
	dim3 gridDim((int) ceil((double)(w-2*r)/(block_size -2*r)), (int) ceil((double)(h-2*r)/(block_size - 2*r)));

    /* Write your code here */
	RGB* d_dst = NULL;
	RGB* d_src = NULL;

	hipMalloc( (void**)&d_dst, sizeof(RGB)*w*h);
	hipMalloc( (void**)&d_src, sizeof(RGB)*w*h);
	hipMemcpy( d_src, (RGB*)h_src, w*h*3, hipMemcpyHostToDevice); 

	//call kernel
	printf("[%d %d %d %d]\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

	cudaPrintfInit();
	mfKernel<<< gridDim, blockDim>>>( d_src, d_dst, r, w, h); 
	FILE *cuprintout;
	cuprintout=fopen("cudadebugprint.txt","w");
	cudaPrintfDisplay(cuprintout, false);
	cudaPrintfEnd();
	fclose(cuprintout);


	hipMemcpy( h_dst, (uchar*)d_dst, w*h*3, hipMemcpyDeviceToHost);

	hipFree(d_dst); 
	hipFree(d_src);
}


__global__ void mfKernel(const RGB* src, RGB* dst, int r, int w, int h)
{
	
	//const int px=threadIdx.x+r;
	//const int py=blockIdx.x+r;
	const int px = blockIdx.x * (blockDim.x -2*r) + threadIdx.x;
	const int py = blockIdx.y * (blockDim.y -2*r) + threadIdx.y;
	RGB pixels[9];
	const int pixelamount=9;
	int i,j;
	int par1=3, par2=3, par3=3, par4=3;

	//create shared memory and copy device_memory into it
	__shared__ RGB subpic[16 * 16];
	if(px<w && py<h)
		subpic[threadIdx.y * blockDim.x + threadIdx.x] = src[ py * w + px];

	//if(blockIdx.x<par1 && blockIdx.y<par2 && threadIdx.x<par3 && threadIdx.y<par4)
	//	cuPrintf("(%d %d %d %d) %d %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, px, py);

	__syncthreads();
	
	//write to pixels
	if(px>=r && px<w-r && py>=r && py<h-r && threadIdx.x>=r && threadIdx.x < blockDim.x-r && threadIdx.y>=r && threadIdx.y < blockDim.y-r)
	{
		i=0;
		for (int dy = -r; dy <= r; ++dy)
		{
			for (int dx = -r; dx <= r; ++dx)
			{        
				pixels[i++] = subpic[ (threadIdx.x+dx) + (threadIdx.y+dy) * blockDim.x];
			}
		}
		//bubble sort, arrange from big to small
		for(i=pixelamount-1; i>0; i--)
		{
			for(j=0; j<i; j++)
			{
				if( computeLuminance(pixels[j].R, pixels[j].G, pixels[j].B) < computeLuminance(pixels[j+1].R, pixels[j+1].G, pixels[j+1].B))
				{
					RGB temp;
					temp=pixels[j];
					pixels[j]=pixels[j+1];
					pixels[j+1]=temp;
				}
			}
		}
		
	}

	if(px>=r && px<w-r && py>=r && py<h-r && threadIdx.x>=r && threadIdx.x < blockDim.x-r && threadIdx.y>=r && threadIdx.y < blockDim.y-r)
		dst[py * w + px] = pixels[((2*r+1)*(2*r+1)-1)/2]; // copy median to output
}